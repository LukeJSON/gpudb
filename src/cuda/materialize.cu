#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "../include/common.h"
#include "../include/schema.h"
#include "../include/gpuCudaLib.h"

__global__ static void materialize(char ** content,  int colNum, int *attrSize, long tupleNum, int tupleSize, char *result){
	int startIndex = blockIdx.x*blockDim.x + threadIdx.x;
	
        int stride = blockDim.x * gridDim.x;

	for(long i=startIndex;i<tupleNum;i+=stride){
		int offset = 0;
		for(int j=0;j<colNum;j++){
			int aSize = attrSize[j];
			memcpy(result+i*tupleSize + offset, content[j]+ i*aSize, aSize);
			offset += aSize;
		}
	}
}

void * materializeCol(struct materializeNode * mn, struct statistic * pp){
	struct tableNode *tn = mn->table;
	char * res, * gpuResult;
	char **gpuContent, **column;
	long size = tn->tupleNum * tn->tupleSize;
	int * gpuAttrSize;

	struct timespec start,end;
	float gpuTime;
	hipEvent_t startGPU, stopGPU;
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);

	clock_gettime(CLOCK_REALTIME,&start);

	column = (char **) malloc(sizeof(char *) * tn->totalAttr);
	if(!column){
		printf("Failed to allocated host memory in MergeCol\n");
		exit(-1);
	}
	
	CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuContent, sizeof(char *) * tn->totalAttr));

	res = (char *) malloc(size);
	CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuResult, size));
	CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuAttrSize,sizeof(int) * tn->totalAttr));

	for(int i=0;i<tn->totalAttr;i++){
		if(tn->dataPos[i] == MEM){
			CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&column[i], tn->tupleNum*tn->attrSize[i]));
			CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[i], tn->content[i], tn->tupleNum *tn->attrSize[i], hipMemcpyHostToDevice));
			CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuContent[i], &column[i], sizeof(char *), hipMemcpyHostToDevice));
		}else if(tn->dataPos[i] == GPU){
			CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuContent[i], &tn->content[i], sizeof(char *), hipMemcpyHostToDevice));
		}
	}

	CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuAttrSize, tn->attrSize, sizeof(int) * tn->totalAttr, hipMemcpyHostToDevice));

	dim3 grid(512);
	dim3 block(128);

	hipEventRecord(startGPU,0);
	materialize<<<grid,block>>> (gpuContent, tn->totalAttr, gpuAttrSize, tn->tupleNum, tn->tupleSize, gpuResult);
	hipDeviceSynchronize();

	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(stopGPU,0));
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&gpuTime,startGPU,stopGPU);

	pp->kernel += gpuTime;

	CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(res, gpuResult, size, hipMemcpyDeviceToHost));

	for(int i=0;i<tn->totalAttr;i++){
		if(tn->dataPos[i] == MEM){
			CUDA_SAFE_CALL_NO_SYNC(hipFree(column[i]));
		}
	}

	free(column);

	CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuContent));
	CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuAttrSize));
	CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuResult));

	clock_gettime(CLOCK_REALTIME,&end);
	double timeE = (end.tv_sec -  start.tv_sec)* BILLION + end.tv_nsec - start.tv_nsec;
	pp->total += timeE/(1000*1000) ;
	return res;
}
