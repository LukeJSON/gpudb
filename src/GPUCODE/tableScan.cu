#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <string.h>
#include <unistd.h>
#include <time.h>
#include "scanImpl.cu"
#include "common.h"
#include "gpulib.h"

__device__ static inline int stringCmp(char* buf1, char *buf2, int size){
	int i;
	int res = 0;
	for(i=0;i<size;i++){
		if(buf1[i] > buf2[i]){
			res = 1;
			break;
		}else if (buf1[i] < buf2[i]){
			res = -1;
			break;
		}
		if(buf1[i] == 0 && buf2[i] == 0)
			break;
	}
	return res;
}

__device__ static inline int testCon(char *buf1, char* buf2, int size, int type, int rel){
	int res = 1;
	if (type == INT){
		if(rel == EQ){
			res = ( *((int*)buf1) == *(((int*)buf2)) );
		}else if (rel == GTH){
			res = ( *((int*)buf1) > *(((int*)buf2)) );
		}else if (rel == LTH){
			res = ( *((int*)buf1) < *(((int*)buf2)) );
		}else if (rel == GEQ){
			res = ( *((int*)buf1) >= *(((int*)buf2)) );
		}else if (rel == LEQ){
			res = ( *((int*)buf1) <= *(((int*)buf2)) );
		}

	}else if (type == FLOAT){
		if(rel == EQ){
			res = ( *((float*)buf1) == *(((float*)buf2)) );
		}else if (rel == GTH){
			res = ( *((float*)buf1) > *(((float*)buf2)) );
		}else if (rel == LTH){
			res = ( *((float*)buf1) < *(((float*)buf2)) );
		}else if (rel == GEQ){
			res = ( *((float*)buf1) >= *(((float*)buf2)) );
		}else if (rel == LEQ){
			res = ( *((float*)buf1) <= *(((float*)buf2)) );
		}

	}else{
		int tmp = stringCmp(buf1,buf2,size);
		if(rel == EQ){
			res = (tmp == 0);
		}else if (rel == GTH){
			res = (tmp > 0);
		}else if (rel == LTH){
			res = (tmp < 0);
		}else if (rel == GEQ){
			res = (tmp >= 0);
		}else if (rel == LEQ){
			res = (tmp <= 0);
		}
	}
	return res;
}


__global__ static void transform_dict_filter_and(int * dictFilter, char *fact, long tupleNum, int dNum,  int * filter, int byteNum){

	int stride = blockDim.x * gridDim.x;
	int offset = blockIdx.x*blockDim.x + threadIdx.x;

	int numInt = (tupleNum * byteNum +sizeof(int) - 1) / sizeof(int) ; 

	for(long i=offset; i<numInt; i += stride){
		int tmp = ((int *)fact)[i];
		unsigned long bit = 1;

		for(int j=0; j< sizeof(int)/byteNum; j++){
			int t = (bit << ((j+1)*byteNum*8)) -1 - ((1<<(j*byteNum*8))-1);
			int fkey = (tmp & t)>> (j*byteNum*8) ;
			filter[i* sizeof(int)/byteNum + j] &= dictFilter[fkey];
		}
	}
}

__global__ static void transform_dict_filter_or(int * dictFilter, char *fact, long tupleNum, int dNum,  int * filter,int byteNum){

	int stride = blockDim.x * gridDim.x;
	int offset = blockIdx.x*blockDim.x + threadIdx.x;

	int numInt = (tupleNum * byteNum +sizeof(int) - 1) / sizeof(int) ;

	for(long i=offset; i<numInt; i += stride){
		int tmp = ((int *)fact)[i];
		unsigned long bit = 1;

		for(int j=0; j< sizeof(int)/byteNum; j++){
			int t = (bit << ((j+1)*byteNum*8)) -1 - ((1<<(j*byteNum*8))-1);
			int fkey = (tmp & t)>> (j*byteNum*8) ;
			filter[i* sizeof(int)/byteNum + j] |= dictFilter[fkey];
		}
	}
}

__global__ static void genScanFilter_dict_or(char *col, int colSize, int colType, int dNum, struct whereExp *where, int *dfilter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	struct dictHeader *dheader = (struct dictHeader *) col;

	for(int i=tid;i<dNum;i+=stride){
		int fkey = dheader->hash[i];
		con = testCon((char *)&fkey,where->content,colSize,colType,where->relation);
		dfilter[i] |= con;
	}
}

__global__ static void genScanFilter_dict_and(char *col, int colSize, int colType, int dNum, struct whereExp *where, int *dfilter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	struct dictHeader *dheader = (struct dictHeader *) col;

	for(int i=tid;i<dNum;i+=stride){
		int fkey = dheader->hash[i];
		con = testCon((char *)&fkey,where->content,colSize,colType,where->relation);
		dfilter[i] &= con;
	}
}

__global__ static void genScanFilter_rle(char *col, int colSize, int colType, long tupleNum, long tupleOffset, struct whereExp *where, int andOr, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	struct rleHeader *rheader = (struct rleHeader *) col;
	int dNum = rheader->dictNum;

	for(int i = tid; i<dNum; i += stride){
		int fkey = ((int *)(col+sizeof(struct rleHeader)))[i];
		int fcount = ((int *)(col+sizeof(struct rleHeader)))[i + dNum];
		int fpos = ((int *)(col+sizeof(struct rleHeader)))[i + 2*dNum];

		if((fcount + fpos) < tupleOffset)
			continue;

		if(fpos >= (tupleOffset + tupleNum))
			break;

		con = testCon((char *)&fkey,where->content,colSize,colType,where->relation);
	
		if(fpos < tupleOffset){
			int tcount = fcount + fpos - tupleOffset;
			if(tcount > tupleNum)
				tcount = tupleNum;
			for(int k=0;k<tcount;k++){
				if(andOr == AND)
					filter[k] &= con;
				else
					filter[k] |= con;
			}

		}else if((fpos + fcount) > (tupleOffset + tupleNum)){
			int tcount = tupleOffset + tupleNum - fpos ;
			for(int k=0;k<tcount;k++){
				if(andOr == AND)
					filter[fpos+k-tupleOffset] &= con;
				else
					filter[fpos+k-tupleOffset] |= con;
			}
		}else{
			for(int k=0;k<fcount;k++){
				if(andOr == AND)
					filter[fpos+k-tupleOffset] &= con;
				else
					filter[fpos+k-tupleOffset] |= con;
			}

		}
	}
}


__global__ static void genScanFilter_and(char *col, int colSize, int  colType, long tupleNum, struct whereExp * where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = testCon(col+colSize*i,where->content,colSize,colType,where->relation);
		filter[i] &= con;
	}
}

__global__ static void genScanFilter_and_int_eq(char *col, long tupleNum, int where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((int*)col)[i] == where; 
		filter[i] &= con;
	}
}

__global__ static void genScanFilter_and_int_geq(char *col, long tupleNum, int where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((int*)col)[i] >= where; 
		filter[i] &= con;
	}
}

__global__ static void genScanFilter_and_int_leq(char *col, long tupleNum, int where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((int*)col)[i] <= where; 
		filter[i] &= con;
	}
}

__global__ static void genScanFilter_and_int_gth(char *col, long tupleNum, int where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((int*)col)[i] > where; 
		filter[i] &= con;
	}
}

__global__ static void genScanFilter_and_int_lth(char *col, long tupleNum, int where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((int*)col)[i] < where;
		filter[i] &= con;
	}
}

__global__ static void genScanFilter_or(char *col, int colSize, int  colType, long tupleNum, struct whereExp * where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;
	int rel = where->relation;

	for(long i = tid; i<tupleNum;i+=stride){
		con = testCon(col+colSize*i,where->content,colSize,colType, rel);
		filter[i] |= con;
	}
}

__global__ static void genScanFilter_or_int_eq(char *col, long tupleNum, int where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((int*)col)[i] == where; 
		filter[i] |= con;
	}
}
__global__ static void genScanFilter_or_float_eq(char *col, long tupleNum, float where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((float*)col)[i] == where; 
		filter[i] |= con;
	}
}
__global__ static void genScanFilter_or_int_gth(char *col, long tupleNum, int where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((int*)col)[i] > where; 
		filter[i] |= con;
	}
}

__global__ static void genScanFilter_or_float_gth(char *col, long tupleNum, float where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((float*)col)[i] > where; 
		filter[i] |= con;
	}
}
__global__ static void genScanFilter_or_int_lth(char *col, long tupleNum, int where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((int*)col)[i] < where; 
		filter[i] |= con;
	}
}
__global__ static void genScanFilter_or_float_lth(char *col, long tupleNum, float where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((float*)col)[i] < where; 
		filter[i] |= con;
	}
}
__global__ static void genScanFilter_or_int_geq(char *col, long tupleNum, int where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((int*)col)[i] >= where;
		filter[i] |= con;
	}
}
__global__ static void genScanFilter_or_float_geq(char *col, long tupleNum, float where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((float*)col)[i] >= where;
		filter[i] |= con;
	}
}
__global__ static void genScanFilter_or_int_leq(char *col, long tupleNum, int where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((int*)col)[i] <= where; 
		filter[i] |= con;
	}
}
__global__ static void genScanFilter_or_float_leq(char *col, long tupleNum, float where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con;

	for(long i = tid; i<tupleNum;i+=stride){
		con = ((float*)col)[i] <= where; 
		filter[i] |= con;
	}
}
__global__ static void genScanFilter(char **col, int colNum, long tupleNum, int *rel, int * where, int * filter){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int con = 1;

	for(long i = tid; i<tupleNum;i+=stride){

		for(int j=0;j<colNum;j++){
			int value = ((int *)(col[j]))[i];
			con &= testCon((char*)&value, (char*)&where[j],sizeof(int), INT, rel[j]);
		}
		filter[i] = con;
	}
}


__global__ static void countScanNum(int *filter, long tupleNum, int * count){
	int stride = blockDim.x * gridDim.x;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int localCount = 0;

	for(long i = tid; i<tupleNum; i += stride){
		localCount += filter[i];
	}

	count[tid] = localCount;

}


__global__ static void scan_dict_other(char *col, char * dict, int byteNum,int colSize, long tupleNum, int *psum, long resultNum, int * filter, char * result){

        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	struct dictHeader *dheader = (struct dictHeader*)dict;
	int pos = psum[tid] * colSize;

	for(long i = tid; i<tupleNum; i+= stride){
		if(filter[i] == 1){
			int key = 0;
			memcpy(&key, col + sizeof(struct dictHeader) + i* dheader->bitNum/8, dheader->bitNum/8);
			memcpy(result+pos,&dheader->hash[key],colSize);
			pos += colSize;
		}
	}
}

__global__ static void scan_dict_int(char *col, char * dict,int byteNum,int colSize, long tupleNum, int *psum, long resultNum, int * filter, char * result){

        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int localCount = psum[tid]; 
	struct dictHeader *dheader = (struct dictHeader*)dict;

	for(long i = tid; i<tupleNum; i+= stride){
		if(filter[i] == 1){
			int key = 0;
			memcpy(&key, col + i*byteNum, byteNum);
			((int *)result)[localCount] = dheader->hash[key];
			localCount ++;
		}
	}	
}

__global__ static void scan_other(char *col, int colSize, long tupleNum, int *psum, long resultNum, int * filter, char * result){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int pos = psum[tid]  * colSize;

	for(long i = tid; i<tupleNum;i+=stride){
		
		if(filter[i] == 1){
			memcpy(result+pos,col+i*colSize,colSize);
			pos += colSize;
		}
	}
}

__global__ static void scan_int(char *col, int colSize, long tupleNum, int *psum, long resultNum, int * filter, char * result){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int localCount = psum[tid] ; 

	for(long i = tid; i<tupleNum;i+=stride){
		
		if(filter[i] == 1){
			((int*)result)[localCount] = ((int*)col)[i];
			localCount ++;
		}
	}
}

__global__ static void scan_all(char **col, int colNum, long tupleNum, int *psum, long resultNum, int * filter, char ** result){
        int stride = blockDim.x * gridDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int localCount = psum[tid] ; 

	for(long i = tid; i<tupleNum;i+=stride){
		
		if(filter[i] == 1){
			for(int j=0;j<colNum;j++){
				((int*)(result[j]))[localCount] = ((int*)(col[j]))[i];
			}
			localCount ++;
		}
	}
}

__global__ void static unpack_rle(char * fact, char * rle, long tupleNum, long tupleOffset, int dNum){

	int offset = blockIdx.x*blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(int i=offset; i<dNum; i+=stride){

		int fvalue = ((int *)(fact+sizeof(struct rleHeader)))[i];
		int fcount = ((int *)(fact+sizeof(struct rleHeader)))[i + dNum];
		int fpos = ((int *)(fact+sizeof(struct rleHeader)))[i + 2*dNum];

		if((fcount + fpos) < tupleOffset)
			continue;

		if(fpos >= (tupleOffset + tupleNum))
			break;

		if(fpos < tupleOffset){
			int tcount = fcount + fpos - tupleOffset;
			if(tcount > tupleNum)
				tcount = tupleNum;
			for(int k=0;k<tcount;k++){
				((int*)rle)[k] = fvalue;
			}

		}else if ((fpos + fcount) > (tupleOffset + tupleNum)){
			int tcount = tupleNum  + tupleOffset - fpos;
			for(int k=0;k<tcount;k++){
				((int*)rle)[fpos-tupleOffset + k] = fvalue;
			}

		}else{
			for(int k=0;k<fcount;k++){
				((int*)rle)[fpos-tupleOffset + k] = fvalue;
			}
		}
	}
}

/*
 * tableScan Prerequisites:
 *	1. the input data can be fit into GPU device memory
 *	2. input data are stored in host memory
 * 
 * Input:
 *	sn: contains the data to be scanned and the predicate information
 *	pp: records statistics such kernel execution time and PCIe transfer time 
 *
 * Output:
 *	A new table node
 */

struct tableNode * tableScan(struct scanNode *sn, struct statistic *pp){

	struct tableNode *res = NULL;

	res = (struct tableNode *) malloc(sizeof(struct tableNode));

	res->totalAttr = sn->tn->totalAttr;
	res->tupleSize = sn->tn->tupleSize;

	res->attrType = (int *) malloc(sizeof(int) * res->totalAttr);
	res->attrSize = (int *) malloc(sizeof(int) * res->totalAttr);
	res->attrTotalSize = (int *) malloc(sizeof(int) * res->totalAttr);
	res->attrIndex = (int *) malloc(sizeof(int) * res->totalAttr);
	res->dataPos = (int *) malloc(sizeof(int) * res->totalAttr);
	res->dataFormat = (int *) malloc(sizeof(int) * res->totalAttr);
	res->content = (char **) malloc(sizeof(char *) * res->totalAttr);

	memcpy(res->attrType, sn->tn->attrType, sizeof(int) * res->totalAttr);
	memcpy(res->attrSize, sn->tn->attrSize, sizeof(int) * res->totalAttr);

	char ** column;
	int * gpuCount;
	int * gpuFilter;
	int * gpuPsum;

	dim3 grid(1024);
	dim3 block(256);

	int blockNum = sn->tn->tupleNum / block.x + 1;

	if(blockNum<1024)
		grid = blockNum;

	int threadNum = grid.x * block.x;
	long totalTupleNum = sn->tn->tupleNum;
	int attrNum;

	attrNum = sn->whereAttrNum;
	column = (char **) malloc(attrNum * sizeof(char *));

	int * whereFree = (int *)malloc(attrNum * sizeof(int));
	int * colWherePos = (int *)malloc(sn->tn->totalAttr * sizeof(int));


	if(!column){
		printf("Failed to allocate host memory\n");
		exit(-1);
	}

	for(int i=0;i<sn->tn->totalAttr;i++)
		colWherePos[i] = -1;

	for(int i=0;i<attrNum;i++){
		whereFree[i] = 1;
		for(int j=0;j<sn->tn->totalAttr;j++){
			if(sn->whereIndex[i] == sn->tn->attrIndex[j]){
				whereFree[i] = -1;
				colWherePos[j] = i;
			}
		}
	}

	int count, *gpuTotalCount;
	CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuFilter,sizeof(int) * totalTupleNum));
	CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuPsum,sizeof(int)*threadNum));
	CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuCount,sizeof(int)*threadNum));

	CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpuPsum,0,sizeof(int) * threadNum));
	CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuTotalCount, sizeof(int)));
	CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpuTotalCount, 0 ,sizeof(int)));
	CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpuFilter,0,sizeof(int) * totalTupleNum));

	assert(sn->hasWhere !=0);
	assert(sn->filter != NULL);

	struct whereCondition *where = sn->filter;

	if(0){

		char ** gpuColumn;
		int * gpuWhere, *cpuWhere;
		int * gpuRel, *cpuRel;

		CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuColumn,sizeof(char *) *where->expNum));
		CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuWhere, sizeof(int) * where->expNum));
		CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuRel, sizeof(int) * where->expNum));

		cpuWhere = (int*)malloc(sizeof(int)* where->expNum);
		cpuRel = (int*)malloc(sizeof(int)* where->expNum);

		int index, prev = -1;
		for(int i=0;i<where->expNum;i++){
			index = where->exp[i].index;
			cpuWhere[i] = *(int *) (where->exp[i].content);
			cpuRel[i] = where->exp[i].relation;
			if(prev != index){
				CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &column[index] , sn->whereSize[index]));
				CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[index], sn->content[index], sn->whereSize[index], hipMemcpyHostToDevice));
				prev = index;
			}
			CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuColumn[i],&column[index], sizeof(char*), hipMemcpyHostToDevice));
		}

		CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuWhere,cpuWhere, sizeof(int)* where->expNum, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuRel,cpuRel, sizeof(int)* where->expNum, hipMemcpyHostToDevice));

		genScanFilter<<<grid,block>>>(gpuColumn, where->expNum,totalTupleNum, gpuRel,gpuWhere,gpuFilter);


	}else{

		struct whereExp * gpuExp;
		CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuExp, sizeof(struct whereExp)));
		CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuExp, &where->exp[0], sizeof(struct whereExp), hipMemcpyHostToDevice));

		int index = where->exp[0].index;
		int prev = index;
		int format = sn->whereFormat[index];

		int prevFormat = format;
		int dNum;
		int byteNum;

		int *gpuDictFilter;

		if(sn->wherePos[index] == MEM || sn->wherePos[index] == PINNED)
			CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &column[index], sn->whereSize[index]));

		if(format == UNCOMPRESSED){
			if(sn->wherePos[index] == MEM || sn->wherePos[index] == PINNED)
				CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[index], sn->content[index], sn->whereSize[index], hipMemcpyHostToDevice));
			else if (sn->wherePos[index] == UVA)
				column[index] = sn->content[index];

			if(sn->whereAttrType[index] == INT){
				int rel = where->exp[0].relation;
				int whereValue = *((int*) where->exp[0].content);
				if(rel==EQ)
					genScanFilter_or_int_eq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
				else if(rel == GTH)
					genScanFilter_or_int_gth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
				else if(rel == LTH)
					genScanFilter_or_int_lth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
				else if(rel == GEQ)
					genScanFilter_or_int_geq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
				else if (rel == LEQ)
					genScanFilter_or_int_leq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);

			}else if (sn->whereAttrType[index] == FLOAT){
				int rel = where->exp[0].relation;
				float whereValue = *((int*) where->exp[0].content);
				if(rel==EQ)
					genScanFilter_or_float_eq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
				else if(rel == GTH)
					genScanFilter_or_float_gth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
				else if(rel == LTH)
					genScanFilter_or_float_lth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
				else if(rel == GEQ)
					genScanFilter_or_float_geq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
				else if (rel == LEQ)
					genScanFilter_or_float_leq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);

			}else
				genScanFilter_or<<<grid,block>>>(column[index],sn->whereAttrSize[index],sn->whereAttrType[index], totalTupleNum, gpuExp, gpuFilter);

		}else if(format == DICT){
			struct dictHeader * dheader = (struct dictHeader *)sn->content[index];
			dNum = dheader->dictNum;
			byteNum = dheader->bitNum/8;

			char * gpuDictHeader;
			CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuDictHeader,sizeof(struct dictHeader)));
			CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuDictHeader,dheader,sizeof(struct dictHeader), hipMemcpyHostToDevice));

			if(sn->wherePos[index] == MEM || sn->wherePos[index] == PINNED)
				CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[index], sn->content[index]+sizeof(struct dictHeader), sn->whereSize[index]-sizeof(struct dictHeader), hipMemcpyHostToDevice));
			else if (sn->wherePos[index] == UVA)
				column[index] = sn->content[index] + sizeof(struct dictHeader);

			CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuDictFilter, dNum * sizeof(int)));
			CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpuDictFilter, 0 ,dNum * sizeof(int)));

			genScanFilter_dict_or<<<grid,block>>>(gpuDictHeader,sn->whereAttrSize[index],sn->whereAttrType[index],dNum, gpuExp,gpuDictFilter);
			CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

			CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuDictHeader));

		}else if(format == RLE){

			if(sn->wherePos[index] == MEM || sn->wherePos[index] == PINNED)
				CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[index], sn->content[index], sn->whereSize[index], hipMemcpyHostToDevice));
			else if (sn->wherePos[index] == UVA)
				column[index] = sn->content[index];

			genScanFilter_rle<<<grid,block>>>(column[index],sn->whereAttrSize[index],sn->whereAttrType[index], totalTupleNum, sn->offset,gpuExp, where->andOr, gpuFilter);
		}

		CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

		int dictFilter = 0;
		int dictFinal = OR;

		for(int i=1;i<where->expNum;i++){
			index = where->exp[i].index;
			format = sn->whereFormat[index];

			CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuExp, &where->exp[i], sizeof(struct whereExp), hipMemcpyHostToDevice));

			if(prev != index){
				if(prevFormat == DICT){
					if(dictFinal == OR)
						transform_dict_filter_or<<<grid,block>>>(gpuDictFilter, column[prev], totalTupleNum, dNum, gpuFilter,byteNum);
					else
						transform_dict_filter_and<<<grid,block>>>(gpuDictFilter, column[prev], totalTupleNum, dNum, gpuFilter,byteNum);

					CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());
					CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuDictFilter));
					dictFinal = where->andOr;
				}

				if(whereFree[prev] == 1 && (sn->wherePos[prev] == MEM || sn->wherePos[prev] == PINNED))
					CUDA_SAFE_CALL_NO_SYNC(hipFree(column[prev]));

				if(sn->wherePos[index] == MEM || sn->wherePos[index] == PINNED)
					CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &column[index] , sn->whereSize[index]));

				if(format == DICT){
					if(sn->wherePos[index] == MEM || sn->wherePos[index] == PINNED)
						CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[index], sn->content[index]+sizeof(struct dictHeader), sn->whereSize[index]-sizeof(struct dictHeader), hipMemcpyHostToDevice));
					else if (sn->wherePos[index] == UVA)
						column[index] = sn->content[index] + sizeof(struct dictHeader);

					struct dictHeader * dheader = (struct dictHeader *)sn->content[index];
					dNum = dheader->dictNum;
					byteNum = dheader->bitNum/8;

					char * gpuDictHeader;
					CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuDictHeader,sizeof(struct dictHeader)));
					CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuDictHeader,dheader,sizeof(struct dictHeader), hipMemcpyHostToDevice));
					CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuDictFilter, dNum * sizeof(int)));
					CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpuDictFilter, 0 ,dNum * sizeof(int)));

					genScanFilter_dict_or<<<grid,block>>>(gpuDictHeader,sn->whereAttrSize[index],sn->whereAttrType[index],dNum, gpuExp,gpuDictFilter);
					dictFilter= -1;
					CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuDictHeader));

				}else{
					if(sn->wherePos[index] == MEM || sn->wherePos[index] == PINNED)
						CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[index], sn->content[index], sn->whereSize[index], hipMemcpyHostToDevice));
					else if (sn->wherePos[index] == UVA)
						column[index] = sn->content[index];
				}

				prev = index;
				prevFormat = format;
			}


			if(format == UNCOMPRESSED){
				if(sn->whereAttrType[index] == INT){
					if(where->andOr == AND){
						int rel = where->exp[i].relation;
						int whereValue = *((int*) where->exp[i].content);
						if(rel==EQ)
							genScanFilter_and_int_eq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == GTH)
							genScanFilter_and_int_gth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == LTH)
							genScanFilter_and_int_lth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == GEQ)
							genScanFilter_and_int_geq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if (rel == LEQ)
							genScanFilter_and_int_leq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
					}else{
						int rel = where->exp[i].relation;
						int whereValue = *((int*) where->exp[i].content);
						if(rel==EQ)
							genScanFilter_or_int_eq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == GTH)
							genScanFilter_or_int_gth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == LTH)
							genScanFilter_or_int_lth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == GEQ)
							genScanFilter_or_int_geq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if (rel == LEQ)
							genScanFilter_or_int_leq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
					}

				} else if (sn->whereAttrType[index] == FLOAT){
					if(where->andOr == AND){
						int rel = where->exp[i].relation;
						float whereValue = *((int*) where->exp[i].content);
						if(rel==EQ)
							genScanFilter_and_float_eq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == GTH)
							genScanFilter_and_float_gth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == LTH)
							genScanFilter_and_float_lth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == GEQ)
							genScanFilter_and_float_geq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if (rel == LEQ)
							genScanFilter_and_float_leq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
					}else{
						int rel = where->exp[i].relation;
						float whereValue = *((int*) where->exp[i].content);
						if(rel==EQ)
							genScanFilter_or_float_eq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == GTH)
							genScanFilter_or_float_gth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == LTH)
							genScanFilter_or_float_lth<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if(rel == GEQ)
							genScanFilter_or_float_geq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
						else if (rel == LEQ)
							genScanFilter_or_float_leq<<<grid,block>>>(column[index],totalTupleNum, whereValue, gpuFilter);
					}
				}else{
					if(where->andOr == AND)
						genScanFilter_and<<<grid,block>>>(column[index],sn->whereAttrSize[index],sn->whereAttrType[index], totalTupleNum, gpuExp, gpuFilter);
					else
						genScanFilter_or<<<grid,block>>>(column[index],sn->whereAttrSize[index],sn->whereAttrType[index], totalTupleNum, gpuExp, gpuFilter);
				}

			}else if(format == DICT){

				struct dictHeader * dheader = (struct dictHeader *)sn->content[index];
				dNum = dheader->dictNum;
				byteNum = dheader->bitNum/8;

				char * gpuDictHeader;
				CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuDictHeader,sizeof(struct dictHeader)));
				CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuDictHeader,dheader,sizeof(struct dictHeader), hipMemcpyHostToDevice));

				if(dictFilter != -1){
					if(where->andOr == AND)
						genScanFilter_dict_and<<<grid,block>>>(gpuDictHeader,sn->whereAttrSize[index],sn->whereAttrType[index],dNum, gpuExp,gpuDictFilter);
					else
						genScanFilter_dict_or<<<grid,block>>>(gpuDictHeader,sn->whereAttrSize[index],sn->whereAttrType[index],dNum, gpuExp,gpuDictFilter);
				}
				dictFilter = 0;

				CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuDictHeader));

			}else if (format == RLE){
				//CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[index], sn->content[index], sn->whereSize[index], hipMemcpyHostToDevice));
				genScanFilter_rle<<<grid,block>>>(column[index],sn->whereAttrSize[index],sn->whereAttrType[index], totalTupleNum, sn->offset,gpuExp, where->andOr, gpuFilter);

			}

			CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());
		}

		if(prevFormat == DICT){
			if(dictFinal == AND)
				transform_dict_filter_and<<<grid,block>>>(gpuDictFilter, column[prev], totalTupleNum, dNum, gpuFilter, byteNum);
			else
				transform_dict_filter_or<<<grid,block>>>(gpuDictFilter, column[prev], totalTupleNum, dNum, gpuFilter, byteNum);
			CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuDictFilter));
		}
	
		if(whereFree[prev] == 1 && (sn->wherePos[prev] == MEM || sn->wherePos[prev] == PINNED))
			CUDA_SAFE_CALL_NO_SYNC(hipFree(column[prev]));

		CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuExp));

	}

	countScanNum<<<grid,block>>>(gpuFilter,totalTupleNum,gpuCount);
	CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

	scanImpl(gpuCount,threadNum,gpuPsum, pp);

	int tmp1, tmp2;

	CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&tmp1, &gpuCount[threadNum-1], sizeof(int), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&tmp2, &gpuPsum[threadNum-1], sizeof(int), hipMemcpyDeviceToHost));

	count = tmp1+tmp2;
	res->tupleNum = count;

	CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuCount));
	CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuTotalCount));


	char **result, **scanCol;

	attrNum = sn->tn->totalAttr;

	scanCol = (char **) malloc(attrNum * sizeof(char *));
	result = (char **) malloc(attrNum * sizeof(char *));
	
	for(int i=0;i<attrNum;i++){

		int pos = colWherePos[i];

		if(pos != -1){
			scanCol[i] = column[pos];
		}else{
			if(sn->tn->dataPos[i] == MEM || sn->tn->dataPos[i] == PINNED)
				CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &scanCol[i] , sn->tn->attrTotalSize[i]));

			if(sn->tn->dataFormat[i] != DICT){
				if(sn->tn->dataPos[i] == MEM || sn->tn->dataPos[i] == PINNED)
					CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(scanCol[i], sn->tn->content[i], sn->tn->attrTotalSize[i], hipMemcpyHostToDevice));
				else
					scanCol[i] = sn->tn->content[i];

			}else{
				if(sn->tn->dataPos[i] == MEM || sn->tn->dataPos[i] == PINNED)
					CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(scanCol[i], sn->tn->content[i]+sizeof(struct dictHeader), sn->tn->attrTotalSize[i]-sizeof(struct dictHeader), hipMemcpyHostToDevice));
				else
					scanCol[i] = sn->tn->content[i] + sizeof(struct dictHeader);
			}
		}

		CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **) &result[i], count * sn->tn->attrSize[i]));
	}

	if(0){

		char ** gpuColumn;
		char ** gpuResult;
		CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuColumn,attrNum * sizeof(char*)));
		CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuResult,attrNum * sizeof(char*)));

		for(int i=0;i<attrNum;i++){
			CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuColumn[i],&scanCol[i],sizeof(char*),hipMemcpyHostToDevice));
			CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuResult[i],&result[i],sizeof(char*),hipMemcpyHostToDevice));
		}

		scan_all<<<grid,block>>>(gpuColumn,attrNum,totalTupleNum,gpuPsum,count,gpuFilter,gpuResult);

	}else{

		for(int i=0; i<attrNum; i++){
			int format = sn->tn->dataFormat[i];
			if(format == UNCOMPRESSED){
				if (sn->tn->attrSize[i] == sizeof(int))
					scan_int<<<grid,block>>>(scanCol[i], sn->tn->attrSize[i], totalTupleNum,gpuPsum,count, gpuFilter,result[i]);
				else
					scan_other<<<grid,block>>>(scanCol[i], sn->tn->attrSize[i], totalTupleNum,gpuPsum,count, gpuFilter,result[i]);

			}else if(format == DICT){
				struct dictHeader * dheader = (struct dictHeader *)sn->tn->content[i];
				int byteNum = dheader->bitNum/8;

				char * gpuDictHeader;
				CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuDictHeader,sizeof(struct dictHeader)));
				CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuDictHeader,dheader,sizeof(struct dictHeader), hipMemcpyHostToDevice));

				if (sn->tn->attrSize[i] == sizeof(int))
					scan_dict_int<<<grid,block>>>(scanCol[i], gpuDictHeader, byteNum,sn->tn->attrSize[i], totalTupleNum,gpuPsum,count, gpuFilter,result[i]);
				else
					scan_dict_other<<<grid,block>>>(scanCol[i], gpuDictHeader,byteNum,sn->tn->attrSize[i], totalTupleNum,gpuPsum,count, gpuFilter,result[i]);

				CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuDictHeader));

			}else if(format == RLE){
				int dNum = (sn->tn->attrTotalSize[i] - sizeof(struct rleHeader))/(3*sizeof(int));
				char * gpuRle;

				CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuRle, totalTupleNum * sizeof(int)));

				unpack_rle<<<grid,block>>>(scanCol[i], gpuRle,totalTupleNum, sn->offset, dNum);

				CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

				scan_int<<<grid,block>>>(gpuRle, sn->tn->attrSize[i], totalTupleNum,gpuPsum,count, gpuFilter,result[i]);

				CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuRle));
			}

		}
	}

	CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

	for(int i=0;i<attrNum;i++){

		if(sn->tn->dataPos[i] == MEM || sn->tn->dataPos[i] == PINNED)
			CUDA_SAFE_CALL_NO_SYNC(hipFree(scanCol[i]));

		int colSize = res->tupleNum * res->attrSize[i];

		res->attrTotalSize[i] = colSize;
		res->dataFormat[i] = UNCOMPRESSED;

		if(sn->keepInGpu == 1){
			res->dataPos[i] = GPU;
			res->content[i] = result[i];
		}else{
			res->dataPos[i] = MEM;
			res->content[i] = (char *)malloc(colSize);
			memset(res->content[i],0,colSize);
			CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(res->content[i],result[i],colSize ,hipMemcpyDeviceToHost));
			CUDA_SAFE_CALL(hipFree(result[i]));
		}
	}

	CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuPsum));
	CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuFilter));

	free(column);
	free(scanCol);
	free(result);

	return res;

}


